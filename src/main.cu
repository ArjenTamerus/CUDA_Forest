/* TODO get tree and leaf array sizes to cut down on GPU memory*/
#include <stdio.h>
#include <string.h>
#include <omp.h>

extern "C" {
  #include "tree.h"
  #include "voxel.h"
  #include "traversal.h"
}
#include "gpu_traversal.h"

//__device__ __constant__ node const_tree[4217];

static float *values;
static short *lindex;

hipError_t copy_const(node *tree, hipStream_t stream);
hipError_t allocate_texture(short *volume);

void gpu_errchk(const char *file, int line, hipError_t cuErr) {
  if(cuErr != hipSuccess)
    printf("EECUDA| Error at %s, %d: %s\n", file, line, hipGetErrorString(cuErr));
}

void copyTreeToGPU(node *tree, node **gpu_tree, leaf *leaves, leaf **gpu_leaves, int treesize, int leafsize) {
  leaf leafptr;
  //float *values;
  //short *index;

  //printf("2.Host->Device:\n %p->%p\tTree\n%p->%p\tLeaf\n", tree, gpu_tree, leaves,
  //    gpu_leaves);

  gpu_errchk(__FILE__, __LINE__, hipMalloc((void**) gpu_tree, treesize *
        sizeof(node)));

  gpu_errchk(__FILE__, __LINE__, hipMalloc((void**) gpu_leaves, sizeof(leaf)));

  gpu_errchk(__FILE__, __LINE__, hipMalloc((void**) &values, leafsize *
        sizeof(float)));

  gpu_errchk(__FILE__, __LINE__, hipMalloc((void**) &lindex, leafsize *
        sizeof(short)));

  leafptr.value = values;
  leafptr.index = lindex;

  gpu_errchk(__FILE__, __LINE__, hipMemcpy(*gpu_leaves, &leafptr, sizeof(leaf),
        hipMemcpyHostToDevice));

  gpu_errchk(__FILE__, __LINE__, hipMemcpy(values, leaves->value, leafsize *
        sizeof(float), hipMemcpyHostToDevice));

  gpu_errchk(__FILE__, __LINE__, hipMemcpy(lindex, leaves->index, leafsize *
        sizeof(short), hipMemcpyHostToDevice));

  gpu_errchk(__FILE__, __LINE__, hipMemcpy(*gpu_tree, tree, treesize *
        sizeof(node), hipMemcpyHostToDevice));

  //printf("3.Host->Device:\n %p->%p\tTree\n%p->%p\tLeaf\n", tree, gpu_tree, leaves,
  //    gpu_leaves);
}

void copyTreeToGPU2(node *tree, node **gpu_tree, leaf *leaves, leaf **gpu_leaves, int treesize, int leafsize, hipStream_t *streams) {
  leaf leafptr;
  //float *values;
  //short *index;

  //printf("2.Host->Device:\n %p->%p\tTree\n%p->%p\tLeaf\n", tree, gpu_tree, leaves,
  //    gpu_leaves);

  /*gpu_errchk(__FILE__, __LINE__, hipMalloc((void**) gpu_tree, treesize *
        sizeof(node)));

  gpu_errchk(__FILE__, __LINE__, hipMalloc((void**) gpu_leaves, sizeof(leaf)));

  gpu_errchk(__FILE__, __LINE__, hipMalloc((void**) &values, leafsize *
        sizeof(float)));

  gpu_errchk(__FILE__, __LINE__, hipMalloc((void**) &index, leafsize *
        sizeof(short)));*/

  leafptr.value = values;
  leafptr.index = lindex;

  gpu_errchk(__FILE__, __LINE__, hipMemcpyAsync(*gpu_leaves, &leafptr, sizeof(leaf),
        hipMemcpyHostToDevice, streams[2]));

  gpu_errchk(__FILE__, __LINE__, hipMemcpyAsync(values, leaves->value, leafsize *
        sizeof(float), hipMemcpyHostToDevice, streams[3]));

  gpu_errchk(__FILE__, __LINE__, hipMemcpyAsync(lindex, leaves->index, leafsize *
        sizeof(short), hipMemcpyHostToDevice, streams[2]));

  gpu_errchk(__FILE__, __LINE__, hipMemcpyAsync(*gpu_tree, tree, treesize *
        sizeof(node), hipMemcpyHostToDevice, streams[3]));

  //printf("3.Host->Device:\n %p->%p\tTree\n%p->%p\tLeaf\n", tree, gpu_tree, leaves,
  //    gpu_leaves);
}
int main(int argc, char **argv) {
  short *volume = NULL, *results = NULL;
  char **classes= NULL;
  node *root = NULL;
  leaf *leaves = NULL;
  node *gpu_root = NULL;
  leaf *gpu_leaves = NULL;
  int classCount;
  int x, y, z;
  int treesize = 4217, leafsize = 16384;

  short *gpu_volume = NULL, *gpu_results = NULL;

  double starttime, endtime;

  volume = parseVolume(&x, &y, &z);

  results = allocate_volume(x, y, z);

  hipDeviceSetCacheConfig(hipFuncCachePreferL1);



  gpu_errchk(__FILE__, __LINE__, hipMalloc((void**) &gpu_volume, x*y*z*sizeof(short)));
  gpu_errchk(__FILE__, __LINE__, hipMalloc((void**) &gpu_results, x*y*z*sizeof(short)));

  gpu_errchk(__FILE__, __LINE__, hipMemcpy(gpu_volume, volume, x*y*z*sizeof(short),
      hipMemcpyHostToDevice));

  printf("DIMS: %d %d %d\n", x, y, z);

  root = parseXMLTree(TREEFILE, &classes, &classCount, &leaves);



  //hipMemcpyToSymbol(HIP_SYMBOL(const_tree), root, 4217 *
  //sizeof(node)));
   //   hipMemcpyHostToDevice));

  //copyTreeToGPU(root, &gpu_root, leaves, &gpu_leaves, treesize, leafsize);

  //starttime = omp_get_wtime();
  //printf("CPU run disabled\n");
  //traverse(root, leaves, volume, results, x, y, z);
  //endtime = omp_get_wtime();

  //printf("Single traversal time, CPU: %f.\n", endtime-starttime);


  /* TODO texture copy */
  //gpu_errchk(__FILE__, __LINE__, allocate_texture(volume));

  dim3 params;
  params.x = 1;
  params.y = 512;
  params.z = 525;
  dim3 params2;
  params2.x = 512;

  hipStream_t streams[4];

  printf("%d ||||||||\n", sizeof(node));

  for(int repeat = 0; repeat < 4; repeat++)
    hipStreamCreate(&streams[repeat]);

  gpu_errchk(__FILE__, __LINE__, copy_const(root, streams[0]));

  starttime = omp_get_wtime();
  //gpu_traverse<<<1,1>>>(gpu_root, gpu_leaves, gpu_volume, gpu_results, x, y, z);
  gpu_traverse<<<params,params2,0,streams[0]>>>(gpu_root, gpu_leaves, gpu_volume, gpu_results, x, y, z);
  for(int repeat = 0; repeat < 3; repeat++) {


    //copyTreeToGPU2(root, &gpu_root, leaves, &gpu_leaves, treesize, leafsize, streams);
    hipDeviceSynchronize();
    gpu_errchk(__FILE__, __LINE__, copy_const(root, streams[0]));
    gpu_traverse<<<params,params2,0,streams[0]>>>(gpu_root, gpu_leaves, gpu_volume, gpu_results, x, y, z);
    gpu_errchk(__FILE__, __LINE__, hipMemcpyAsync(results, gpu_results,
        x*y*z*sizeof(short), hipMemcpyDeviceToHost, streams[1]));



  }
  hipDeviceSynchronize();
  endtime = omp_get_wtime();

  for(int repeat = 0; repeat < 4; repeat++)
    hipStreamDestroy(streams[repeat]);



  /*Copy results back to host*/
  gpu_errchk(__FILE__, __LINE__, hipMemcpy(results, gpu_results,
      x*y*z*sizeof(short), hipMemcpyDeviceToHost));

  printf("Single traversal time, GPU: %f.\n", endtime-starttime);

  if(argc == 3) {
    if(!(strcmp(argv[1], "-csv"))) {
      printf("Printing results to %s.\n", argv[2]);
      toCSV(results, x, y, z, argv[2]);
    }
  }

  gpu_errchk(__FILE__, __LINE__, hipFree(gpu_volume));
  gpu_errchk(__FILE__, __LINE__, hipFree(gpu_results));

  freeVolume(volume, y, z);
  freeVolume(results, y, z);
  classes = freeClasses(classes, classCount);

  free(root);
  free(leaves);

  return 0;
}
