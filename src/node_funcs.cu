#include "hip/hip_runtime.h"
#include <float.h>

#include "gpu_node_funcs.h"
#include "gpu_traversal.h"
extern "C" {
  #include "voxel.h"
}
void gpu_errchk(const char *file, int line, hipError_t cuErr);

#define test(func) gpu_errchk(__FILE__,__LINE__,func)

//__device__ __constant__ node const_tree[4217];
texture<short, 3, hipReadModeElementType> tex;
hipArray *volume_array;

__device__ int gpu_longRangeContext(short *voxels, char *coords, float threshold, int a, int b, int c, int x, int y, int z) {
  //int pos = (c * y * x) + (b * x) + a;
  if(voxels[a] < threshold)
    return LEFT;
  return RIGHT;
}

// TODO
__device__ int gpu_coronal(short voxel, int y_center, int z_center, float threshold) {
  if((voxel - y_center) < threshold)
    return LEFT;
  return RIGHT;
}

#define treenode const_tree[cur]
__global__ /*__launch_bounds__(512, 4)*/ void gpu_traverse(node *tree, leaf *leaves, short *voxels, short *results, int x, int y, int z) {

  node *curr;
  int cur;
  int i, j, k;
  int a, b, c;
  int pos, alt_pos;
  char res;
  int cookie;

  /* XXX i-j-k = z-y-x */

  i = blockIdx.z;
  /*if(blockIdx.y & 3 == 1) {
    j = (blockIdx.y / 4);
    k = threadIdx.x + 128;
  } else if(blockIdx.y & 3 == 2) {
    j = (blockIdx.y / 4);
    k = threadIdx.x + 256;
  } else if(blockIdx.y & 3 == 3) {
    j = (blockIdx.y / 4);
    k = threadIdx.x + 384;
  } else {
    j = (blockIdx.y / 4);
    k = threadIdx.x;
  }*/
  j = blockIdx.y;
  k = threadIdx.x;


  cur = 0;

  pos = (i * x * y) + (j * x) + k;
  while(1) {
    if(treenode.type == LONGRANGECONTEXT) {
      a = (k + treenode.arguments[0]) & (x-1);
      b = (j + treenode.arguments[1]) & (x-1);
      c = i + treenode.arguments[2];

      if(c < 0)
        c += z;
      else if (c >= z)
        c -= z;

      alt_pos = (c * y * x) + (b * x) + a;
      res = voxels[alt_pos] < treenode.threshold ? LEFT : RIGHT;
      cur = treenode.children[res];
    }
    else if(treenode.type == CORONAL) {
      res = (x - (y>>1)) < treenode.threshold ? LEFT : RIGHT;
      cur = treenode.children[res];
    }
    else {
      break;
    }
  }

  results[pos] = treenode.arguments[0]; //leaf number/ID

}

hipError_t copy_const(node *tree, hipStream_t stream) {
  //printf("Host root type: %d\n", tree[0].type);
  return hipMemcpyToSymbolAsync(HIP_SYMBOL(const_tree), tree, 4217 * sizeof(node), 0,
  hipMemcpyHostToDevice, stream);
}

hipError_t allocate_texture(short *volume) {
  tex.addressMode[0] = hipAddressModeWrap;
  tex.addressMode[1] = hipAddressModeWrap;
  tex.addressMode[2] = hipAddressModeWrap;
  tex.filterMode = hipFilterModeLinear;
  tex.normalized = false;

  hipChannelFormatDesc format = {16, 0, 0, 0, hipChannelFormatKindSigned};

  hipExtent extent = {512, 512, 525};

  test(hipMalloc3DArray(&volume_array, &format, extent, 0));

  hipMemcpy3DParms params = {0};

  int pitch = sizeof(float);

  params.srcPtr = make_hipPitchedPtr((void*) volume, pitch, 512*512*525, 1);
  params.dstArray = volume_array;
  params.extent = extent;
  params.kind = hipMemcpyHostToDevice;

  test(hipMemcpy3D(&params));


  //test(hipMemcpyToArray(volume_array, 0, 0, volume, 512*512*525, hipMemcpyHostToDevice));

  return hipBindTextureToArray(tex, volume_array, format);
}
